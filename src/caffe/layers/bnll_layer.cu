#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

using std::max;

namespace caffe {

const float kBNLL_THRESHOLD = 50.;

template <typename Dtype>
__global__ void BNLLForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ?
        in[index] + log(1. + exp(-in[index])) :
        log(1. + exp(in[index]));
  }
}

template <typename Dtype>
void BNLLLayer<Dtype>::NeuronForward_gpu(const Blob<Dtype>& bottom,
    Blob<Dtype>* top) {
  const Dtype* bottom_data = bottom.gpu_data();
  Dtype* top_data = top->mutable_gpu_data();
  const int count = bottom.count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  BNLLForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void BNLLBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype expval = exp(min(in_data[index], Dtype(kBNLL_THRESHOLD)));
    out_diff[index] = in_diff[index] * expval / (expval + 1.);
  }
}

template <typename Dtype>
void BNLLLayer<Dtype>::NeuronBackward_gpu(const Blob<Dtype>& top,
    Blob<Dtype>* bottom) {
  const Dtype* bottom_data = bottom->gpu_data();
  const Dtype* top_diff = top.gpu_diff();
  Dtype* bottom_diff = bottom->mutable_gpu_diff();
  const int count = bottom->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  BNLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff, bottom_data, bottom_diff);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_CLASS(BNLLLayer);


}  // namespace caffe
