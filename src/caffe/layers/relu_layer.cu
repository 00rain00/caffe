#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

using std::max;

namespace caffe {

template <typename Dtype>
__global__ void ReLUForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : 0;
  }
}

template <typename Dtype>
void ReLULayer<Dtype>::NeuronForward_gpu(const Blob<Dtype>& bottom,
    Blob<Dtype>* top) {
  const Dtype* bottom_data = bottom.gpu_data();
  Dtype* top_data = top->mutable_gpu_data();
  const int count = bottom.count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void ReLUBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * (in_data[index] > 0);
  }
}

template <typename Dtype>
void ReLULayer<Dtype>::NeuronBackward_gpu(const Blob<Dtype>& top,
    Blob<Dtype>* bottom) {
  const Dtype* bottom_data = bottom->gpu_data();
  const Dtype* top_diff = top.gpu_diff();
  Dtype* bottom_diff = bottom->mutable_gpu_diff();
  const int count = bottom->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff, bottom_data, bottom_diff);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_CLASS(ReLULayer);


}  // namespace caffe
